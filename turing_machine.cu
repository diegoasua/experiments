#include <iostream>
#include <hip/hip_runtime.h>

#define STATE_INITIAL 0
#define STATE_FINAL 1
#define HALT_STATE -1
#define TAPE_SIZE 100

__global__ void turingMachineSimulator(char *tape, int *state, int *headPos) {
    while (*state != HALT_STATE) {
        char currentSymbol = tape[*headPos];
        switch (*state) {
            case STATE_INITIAL:
                if (currentSymbol == '1') {
                    tape[*headPos] = '0';
                    (*headPos)++;
                    *state = STATE_FINAL;
                } else {
                    *state = HALT_STATE;
                }
                break;
            case STATE_FINAL:
                if (currentSymbol == '0') {
                    tape[*headPos] = '1';
                    (*headPos)--;
                    *state = HALT_STATE;
                } else {
                    *state = HALT_STATE;
                }
                break;
            default:
                *state = HALT_STATE;
        }
    }
}

int main() {
    char h_tape[TAPE_SIZE];
    int h_state = STATE_INITIAL;
    int h_headPos = 0;

    for (int i = 0; i < TAPE_SIZE; i++) {
        h_tape[i] = (i % 2 == 0) ? '1' : '0';
    }

    char *d_tape;
    int *d_state, *d_headPos;
    hipMalloc(&d_tape, TAPE_SIZE * sizeof(char));
    hipMalloc(&d_state, sizeof(int));
    hipMalloc(&d_headPos, sizeof(int));

    hipMemcpy(d_tape, h_tape, TAPE_SIZE * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_state, &h_state, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_headPos, &h_headPos, sizeof(int), hipMemcpyHostToDevice);

    turingMachineSimulator<<<1, 1>>>(d_tape, d_state, d_headPos);

    hipMemcpy(h_tape, d_tape, TAPE_SIZE * sizeof(char), hipMemcpyDeviceToHost);
    hipMemcpy(&h_state, d_state, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_headPos, d_headPos, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Final tape: ";
    for (int i = 0; i < TAPE_SIZE; i++) {
        std::cout << h_tape[i];
    }
    std::cout << std::endl;

    std::cout << "Final state: " << h_state << std::endl;
    std::cout << "Final head position: " << h_headPos << std::endl;

    hipFree(d_tape);
    hipFree(d_state);
    hipFree(d_headPos);

    return 0;
}
